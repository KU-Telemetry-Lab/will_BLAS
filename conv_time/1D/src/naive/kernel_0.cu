#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "kernel_0.h"

__global__ void ConvKernel(int* I, int* F, int* R, int N, int M) {
    // I = input vector of length N
    // F = filter vector of length M
    // R = results vector of length N+M-1

    // determine thread id
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // check boundary of the resulting vector
    if (thread_id >= N+M-1) return;

    int temp = 0;        
    // step through each element in the filter
    for (int i = 0; i < M; i++) {
        // calculate the corresponding index in the input vector
        int inputIdx = thread_id - i;
        // check if the index is within the bounds of the input vector
        if (inputIdx >= 0 && inputIdx < N) {
            temp += I[inputIdx] * F[i];
        }
    }

    // write back results
    R[thread_id] = temp;
}


void Conv(int* I, int* F, int *R, int N, int M) {
    dim3 threadsPerBlock(N+M-1);
    dim3 blocksPerGrid(1);
    if ((N+M-1) > 512){
        threadsPerBlock.x = 512;
        blocksPerGrid.x = ceil(double(N+M-1)/double(threadsPerBlock.x));
    }

    ConvKernel<<<blocksPerGrid, threadsPerBlock>>>(I, F, R, N, M);
}

