#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>

#include "kernel_1.h"

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;


int main(int argc, char **argv)
{
    // initializing timing parameters for recording
    // cpu and gpu speeds
    using clock = std::chrono::steady_clock;
    clock::time_point startTime;
    clock::time_point endTime;
    clock::duration allTime;
    uint64_t memcpy_timeMsec;
    uint64_t memrd_timeMsec;
    uint64_t kernel_timeMsec;
    uint64_t total_timeMsec;
    uint64_t cpu_timeMsec;


    if (argc < 2) {
        printf("usage:  %s <maxtrix dim>\n", argv[0]);
        exit(-1);
    }

    int N;
    N = atoi(argv[1]);


    // initialize host matricies h_A, h_B, and h_C on host
    Matrix h_A;
    h_A.width = N;
    h_A.height = N;
    size_t size_h_A = h_A.height * h_A.width * sizeof(float);

    Matrix h_B;
    h_B.width = N;
    h_B.height = N;
    size_t size_h_B = h_B.height * h_B.width * sizeof(float);

    Matrix h_C;
    h_C.width = N;
    h_C.height = N;
    size_t size_h_C = h_C.height * h_C.width * sizeof(float);


    // allocate memory on host for h_A, h_B, and h_C
    h_A.elements = (float*)malloc(size_h_A);
    h_B.elements = (float*)malloc(size_h_B);
    h_C.elements = (float*)malloc(size_h_C);


    // fill matricies h_A and h_B on host
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            // h_A[i][j]
            h_A.elements[i*N+j] = sin(i);
            h_B.elements[i*N+j] = cos(j);
        }
    }


    // initialize matricies d_A, d_B, and d_C on device
    Matrix d_A;
    d_A.width = h_A.width;
    d_A.height = h_A.height;
    size_t size_d_A = h_A.width * h_A.height * sizeof(float);

    Matrix d_B;
    d_B.width = h_B.width; 
    d_B.height = h_B.height;
    size_t size_d_B = h_B.width * h_B.height * sizeof(float);

    Matrix d_C;
    d_C.width = h_C.width; 
    d_C.height = h_C.height;
    size_t size_d_C = h_C.width * h_C.height * sizeof(float);


    // allocate memory on decive for d_A, d_B, and d_C (result)
    hipMalloc(&d_A.elements, size_d_A);
    hipMalloc(&d_B.elements, size_d_B);
    hipMalloc(&d_C.elements, size_d_C);


    // copy h_A and h_B from host memory to d_A and d_B in device memory (and time it)
    startTime = clock::now();
    hipMemcpy(d_A.elements, h_A.elements, size_d_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B.elements, h_B.elements, size_d_B, hipMemcpyHostToDevice);
    endTime = clock::now();
    allTime = endTime - startTime;

    // memcpy timing results
    memcpy_timeMsec = std::chrono::duration_cast<std::chrono::microseconds>(allTime).count();
    printf("%8lu ms to copy 2 %dx%d matricies from CPU memory to GPU memory.\n", memcpy_timeMsec, N, N);



    // perform matrix multiplication on device (and time it)
    startTime = clock::now();
    MatMult_1(d_A.elements, d_B.elements, d_C.elements, N);
    // wait for all threads to complete
    hipDeviceSynchronize();
    endTime = clock::now();
    allTime = endTime - startTime;

    // kernel timing results
    kernel_timeMsec = std::chrono::duration_cast<std::chrono::microseconds>(allTime).count();
    printf("%8lu ms to perform %dx%d matrix multiply on GPU.\n", kernel_timeMsec, N, N);


    // copy d_C from device memory to h_C in host memory (and time it)
    startTime = clock::now();
    hipMemcpy(h_C.elements, d_C.elements, size_h_C, hipMemcpyDeviceToHost);
    endTime = clock::now();
    allTime = endTime - startTime;

    // memcpy timing results
    memrd_timeMsec = std::chrono::duration_cast<std::chrono::microseconds>(allTime).count();
    printf("%8lu ms to copy 1 %dx%d matrix from GPU memory to CPU memory.\n\n", memrd_timeMsec, N, N);

    total_timeMsec = memcpy_timeMsec + kernel_timeMsec + memrd_timeMsec;
    printf("%8lu ms for full %dx%d matrix multiply and data transfers.\n\n", total_timeMsec, N, N);

    // ERROR CHECKING AND CPU COMPARISON #############################################
    // initialize matrix cpu_C to hold cpu results
    Matrix cpu_C;
    cpu_C.width = N;
    cpu_C.height = N;
    size_t size_cpu_C = cpu_C.height * cpu_C.width * sizeof(float);

    // allocate memory on host for cpu_C
    cpu_C.elements = (float*)malloc(size_cpu_C);

    // performing matrix multiplication on CPU (and time it)
    startTime = clock::now();
    float sum;
    for (int row=0; row<N; row++){
        for (int col=0; col<N; col++){
        sum = 0.f;
        for (int n=0; n<N; n++){
            sum += h_A.elements[row*N+n] * h_B.elements[n*N+col];
        }
        cpu_C.elements[row*N+col] = sum;
        }
    }
    endTime = clock::now();
    allTime = endTime - startTime;
    cpu_timeMsec = std::chrono::duration_cast<std::chrono::microseconds>(allTime).count();
    printf("%8lu ms for full %dx%d matrix multiply on CPU.\n\n\n", cpu_timeMsec, N, N);
    
    // error checking
    int error_count = 0;
    for (int i = 0; i < (N*N); i++){
        //  need to fix found value...
        float epsilon = .0001;
        if (fabs(h_C.elements[i] - cpu_C.elements[i]) > epsilon) {
            error_count += 1;   
            printf("element %d GPU: %f | CPU: %f\n", i, h_C.elements[i], cpu_C.elements[i]);
        }
    }


    printf("Total errors in GPU matrix: %d\n", error_count);


    // free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

    // free host memory
    free(h_A.elements);
    free(h_B.elements);
    free(h_C.elements);
    free(cpu_C.elements);
}