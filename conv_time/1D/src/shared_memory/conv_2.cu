#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <time.h>

#include "kernel_2.h"
#define FILTER_LENGTH 512

typedef struct {
    int* elements;
} Vector;


void validate_result(int *I, int *H, int *R, int N, int M) {
    // I = input vector of length N
    // F = filter vector of length M
    // R = GPU results vector of length N+M-1

    for (int i = 0; i < N + M - 1; i++) {
        int temp = 0;

        // determine the start and end indices for the convolution operation
        int start = i >= M ? i - M + 1 : 0;
        int end = i < N ? i : N - 1;

        for (int j = start; j <= end; j++) {
            if((i-j) >= 0 && (i-j) < M) {
                temp += I[j] * H[(i-j)];
            }
        }
        // ensure temp matches result from GPU
        // printf("%d", R[i]);
        assert(temp == R[i]);
        // if (temp != R[i]) {
        //     printf("temp: %d | R: %d\n", temp, R[i]);
        // }
    }
}


int next_power_two(int x) {
    if ((x && !(x & (x - 1)))) return x;

    int power = 1;
    while (power < x) {
        power <<= 1;
    }
    return power;
}


void print_vector(int* I, int N) {
    printf(" Vector = [");
    for (int i = 0; i < N; i++){
        printf("%d ", I[i]);
    }
    printf("]\n\n");
}


int main(int argc, char **argv) {

    using clock = std::chrono::steady_clock;
    clock::time_point startTime;
    clock::time_point endTime;
    clock::duration allTime;
    uint64_t timeMsec;

    if (argc < 2) {
        printf("usage:  %s <input length>\n", argv[0]);
        exit(-1);
    }


    int N = atoi(argv[1]); // length of input vector
    int M = FILTER_LENGTH; // length of filter vector
    int R = N+FILTER_LENGTH-1; // resulting vector size
    int padded_length = next_power_two(R); // bumping R to the next power of two


    // initialize host vectors h_I (input), h_F (filter), and h_R (result)
    Vector h_I, h_F, h_R;
    size_t size_h = padded_length * sizeof(int);

    // allocate memory on host for h_I, h_F, and h_R
    h_I.elements = (int*)malloc(size_h);
    h_F.elements = (int*)malloc(size_h);
    h_R.elements = (int*)malloc(size_h);


    // fill real vectors of h_I and h_F on host
    if (N >= M) {
        for (int i=0; i<N; i++){
            // h_I.elements[i] = sin(i);
            h_I.elements[i] = 1;
            if (i < M) {
                // h_F.elements[i] = rand() % 10;
                h_F.elements[i] = 1;
            }
        }
        // zero pad remaining values in h_I and h_F
        for (int i = N; i < padded_length; i++) {
            h_I.elements[i] = 0;
            h_F.elements[i] = 0;
        }
    } else {
        printf("Input length N (%d) greater than filter length M (%d)...\n", N, N);
        return 0;
    }

    // initialize device vectors d_I (input) and d_R (result)
    Vector d_I, d_R;
    size_t size_d = padded_length * sizeof(int);

    // allocate memory on device  for d_I and d_R
    hipMalloc(&d_I.elements, size_d);
    hipMalloc(&d_R.elements, size_d);

    startTime = clock::now();

    // copy h_I and h_F from host memory to d_I and d_F in device memory
    hipMemcpy(d_I.elements, h_I.elements, size_d, hipMemcpyHostToDevice);

    // copy the data directly to the symbol (no offset)
    FillConstant(h_F.elements);

    // call 1D convolution kernel helper function
    Conv(d_I.elements, d_R.elements, M, padded_length);
    // wait for all thread blocks to execute
    hipDeviceSynchronize();

    // copy d_R from device memory to h_R in host memory
    hipMemcpy(h_R.elements, d_R.elements, size_h, hipMemcpyDeviceToHost);

    endTime = clock::now();
    allTime = endTime - startTime;
    timeMsec = std::chrono::duration_cast<std::chrono::microseconds>(allTime).count();
    printf("%8lu microsec for 1D convolution on GPU.\n", timeMsec);

    // validate result on host
    validate_result(h_I.elements, h_F.elements, h_R.elements, N, M);

    // free device memory
    hipFree(d_I.elements);
    hipFree(d_R.elements);

    // free host memory
    free(h_I.elements);
    free(h_F.elements);
    free(h_R.elements);
}