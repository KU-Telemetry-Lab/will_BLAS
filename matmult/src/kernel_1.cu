#include "hip/hip_runtime.h"
#include <math.h>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "kernel_1.h"

#define BLOCKSIZE 32

__global__ void MatMultKernel_1(float* A, float* B, float* C, int N) {

    const int x = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const int y = blockIdx.y * BLOCKSIZE + (threadIdx.y % BLOCKSIZE);

    float tmpSum = 0.0;

    if (x < N && y < N) {
        for (int i = 0; i < N; i++) {
            tmpSum += A[x * N + i] * B[i * N + y];
        }
    }
    C[x * N + y] = tmpSum;
}

void MatMult_1(float *A, float *B, float *C, int N){
    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 blocksPerGrid((N + BLOCKSIZE - 1) / BLOCKSIZE, (N + BLOCKSIZE - 1) / BLOCKSIZE);

    MatMultKernel_1<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, N);
}
