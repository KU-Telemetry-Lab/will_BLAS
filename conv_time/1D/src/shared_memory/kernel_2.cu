#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>

#include "kernel_2.h"
#define FILTER_LENGTH 512


__constant__ int F[FILTER_LENGTH];

__global__ void ConvKernel(int* I, int* R, int filter_length, int padded_length) {
    // determine thread id
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // store elements needed to compute output in shared memory
    extern __shared__ int shared[];

     // load elements from input vector to shared, checking boundary
    if (thread_id < padded_length) {
        shared[threadIdx.x] = I[thread_id];
    } else {
        // padding with 0 for threads beyond padded_length
        shared[threadIdx.x] = 0;
    }
    __syncthreads(); // ensure all loads are complete

    // only proceed if thread_id within boundary of result
    if (thread_id >= padded_length) return;

    int temp = 0;        
    // step through each element in the filter
    for (int i = 0; i < filter_length; i++) {
        int shared_id = threadIdx.x - i;
        if (shared_id >= 0 && shared_id < blockDim.x) {
            // value is in shared memory
            temp += shared[shared_id] * F[i];
        } else {
            // value is outside shared memory (global memory access)
            temp += I[thread_id - i] * F[i];
        }
    }
    // write back results
    R[thread_id] = temp;
}


void Conv(int* I, int *R, int filter_length, int padded_length) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (padded_length + threadsPerBlock - 1) / threadsPerBlock;
    size_t sharedMemSize = threadsPerBlock * sizeof(int);

    ConvKernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(I, R, filter_length, padded_length);
}

void FillConstant(int* I) {
    hipMemcpyToSymbol(HIP_SYMBOL(F), I, FILTER_LENGTH * sizeof(int));
}