#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include "kernel_1.h"

#define PI 3.14159265358979323846

using namespace std;

__global__ void PointWiseMultKernel(hipfftComplex* I, hipfftComplex* F, hipfftComplex* R, int N) {
    // determine thread id
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // check boundary of the resulting vector
    if (thread_id >= N) return;

    float a = I[thread_id].x; 
    float b = I[thread_id].y;
    float c = F[thread_id].x; 
    float d = F[thread_id].y;

    R[thread_id].x = ((a * c) - (b * d)) / N; // scale to 1/N
    R[thread_id].y = ((a * d) + (b * c)) / N; // scale to 1/N
}


void PointWiseMult(hipfftComplex* I, hipfftComplex* F, hipfftComplex* R, int N) {
    dim3 threadsPerBlock(N);
    dim3 blocksPerGrid(1);
    if ((N) > 512){
        threadsPerBlock.x = 512;
        blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
    }

    PointWiseMultKernel<<<blocksPerGrid, threadsPerBlock>>>(I, F, R, N);
}


void fft(hipfftComplex* I, int N) {
    if (N <= 1) return;

    hipfftComplex even[N/2];
    hipfftComplex odd[N/2];

    for (int i = 0; i < N / 2; i++) {
        even[i].x = I[2 * i].x;
        even[i].y = I[2 * i].y;
        odd[i].x = I[2 * i + 1].x;
        odd[i].y = I[2 * i + 1].y;
    }

    fft(even, N/2);
    fft(odd, N/2);

    for (int k = 0; k < N / 2; k++) {
        float cos_theta = cos(-2 * PI * k / N);
        float sin_theta = sin(-2 * PI * k / N);

        float t_real = cos_theta * odd[k].x - sin_theta * odd[k].y;
        float t_imag = sin_theta * odd[k].x + cos_theta * odd[k].y;

        I[k].x = even[k].x + t_real;
        I[k].y = even[k].y + t_imag;

        I[k + N/2].x = even[k].x - t_real;
        I[k + N/2].y = even[k].y - t_imag;
    }
}


void ifft(hipfftComplex* I, int N) {
    if (N <= 1) return;

    hipfftComplex even[N/2];
    hipfftComplex odd[N/2];

    for (int i = 0; i < N / 2; i++) {
        even[i].x = I[2 * i].x;
        even[i].y = I[2 * i].y;
        odd[i].x = I[2 * i + 1].x;
        odd[i].y = I[2 * i + 1].y;
    }

    // Recursive calls for even and odd components
    ifft(even, N/2);
    ifft(odd, N/2);

    for (int k = 0; k < N / 2; k++) {
        // Use positive sign for the IFFT exponent
        float cos_theta = cos(2 * PI * k / N);
        float sin_theta = sin(2 * PI * k / N);

        // Combine even and odd components
        float t_real = cos_theta * odd[k].x - sin_theta * odd[k].y;
        float t_imag = sin_theta * odd[k].x + cos_theta * odd[k].y;

        I[k].x = even[k].x + t_real;
        I[k].y = even[k].y + t_imag;

        I[k + N/2].x = even[k].x - t_real;
        I[k + N/2].y = even[k].y - t_imag;
    }
}


int is_power_two(int x) {
    return x && !(x & (x - 1));
}


int next_power_two(int x) {
    if (is_power_two(x)) return x;

    int power = 1;
    while (power < x) {
        power <<= 1;
    }
    return power;
}

void print_complex_vector(hipfftComplex* I, int N) {
    printf("Real Vector = [");
    for (int i = 0; i < N; i++){
        printf("%f ", I[i].x);
    }
    printf("]\n\n");

    printf("Imaginary Vector = [");
    for (int i = 0; i < N; i++){
        printf("%f ", I[i].y);
    }
    printf("]\n\n");
}
