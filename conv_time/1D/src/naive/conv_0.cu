#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <time.h>

#include "kernel_0.h"
#define FILTER_LENGTH 512


typedef struct {
    int length;
    int* elements;
} Vector;


void validate_result(int *I, int *F, int *R, int N, int M) {
    // I = input vector of length N
    // F = filter vector of length M
    // R = GPU results vector of length N+M-1

    for (int i = 0; i < N + M - 1; i++) {
        int temp = 0;

        // determine the start and end indices for the convolution operation
        int start = i >= M ? i - M + 1 : 0;
        int end = i < N ? i : N - 1;

        for (int j = start; j <= end; j++) {
            if((i-j) >= 0 && (i-j) < M) {
                temp += I[j] * F[(i-j)];
            }
        }
        // ensure temp matches result from GPU
        // printf("%d", R[i]);
        assert(temp == R[i]);
    }
}


int main(int argc, char **argv) {

    using clock = std::chrono::steady_clock;
    clock::time_point startTime;
    clock::time_point endTime;
    clock::duration allTime;
    uint64_t timeMsec;

    if (argc < 2) {
        printf("usage:  %s <input length>\n", argv[0]);
        exit(-1);
    }


    int N = atoi(argv[1]); // length of input vector
    int M = FILTER_LENGTH; // length of filter vector
    int R = N+FILTER_LENGTH-1; // resulting vector size


    // initialize host vectors h_I (input), h_F (filter), and h_R (result)
    Vector h_I;
    h_I.length = N;
    size_t size_h_I = h_I.length * sizeof(int);

    Vector h_F;
    h_F.length = M;
    size_t size_h_F = h_F.length * sizeof(int);

    Vector h_R;
    h_R.length = R;
    size_t size_h_R = h_R.length * sizeof(int);


    // allocate memory on host for h_I, h_F, and h_R
    h_I.elements = (int*)malloc(size_h_I);
    h_F.elements = (int*)malloc(size_h_F);
    h_R.elements = (int*)malloc(size_h_R);


    // fill vectors h_I and h_F on host
    if (N >= M) {
        for (int i=0; i<N; i++){
            // h_I.elements[i] = sin(i);
            h_I.elements[i] = 1;
            if (i <= M) {
                // h_F.elements[i] = rand() % 10;
                h_F.elements[i] = 1;
            }
        }
    } else {
        printf("Input length N (%d) greater than filter length M (%d)...\n", N, N);
        return 0;
    }

    // initialize device vectors d_I (input), d_F (filter), and d_R (result)
    Vector d_I;
    d_I.length = N;
    size_t size_d_I = d_I.length * sizeof(int);

    Vector d_F;
    d_F.length = M;
    size_t size_d_F = d_F.length * sizeof(int);
    
    Vector d_R;
    d_R.length = R;
    size_t size_d_R = d_R.length * sizeof(int);

    // allocate memory on device  for d_I, d_F, and d_R
    hipMalloc(&d_I.elements, size_d_I);
    hipMalloc(&d_F.elements, size_d_F);
    hipMalloc(&d_R.elements, size_d_R);

    startTime = clock::now();

    // copy h_I and h_F from host memory to d_I and d_F in device memory
    hipMemcpy(d_I.elements, h_I.elements, size_d_I, hipMemcpyHostToDevice);
    hipMemcpy(d_F.elements, h_F.elements, size_d_F, hipMemcpyHostToDevice);

    // call 1D convolution kernel helper function
    Conv(d_I.elements, d_F.elements, d_R.elements, N, M);
    // wait for all thread blocks to execute
    hipDeviceSynchronize();

    // copy d_R from device memory to h_R in host memory
    hipMemcpy(h_R.elements, d_R.elements, size_h_R, hipMemcpyDeviceToHost);

    endTime = clock::now();
    allTime = endTime - startTime;
    timeMsec = std::chrono::duration_cast<std::chrono::microseconds>(allTime).count();
    printf("%8lu microsec for 1D convolution on GPU.\n", timeMsec);

    // validate result on host
    validate_result(h_I.elements, h_F.elements, h_R.elements, N, M);

    // free device memory
    hipFree(d_I.elements);
    hipFree(d_F.elements);
    hipFree(d_R.elements);

    // free host memory
    free(h_I.elements);
    free(h_F.elements);
    free(h_R.elements);
}