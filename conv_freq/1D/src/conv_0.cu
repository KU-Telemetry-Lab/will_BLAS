#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <math.h>
#include <chrono>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include "kernel_1.h"


typedef struct {
    int length;
    hipfftComplex* elements;
} VectorComplex;


int main(int argc, char **argv) {

    if (argc < 2) {
        printf("usage:  %s <y_length> <x_length>\n", argv[0]);
        exit(-1);
    }


    using clock = std::chrono::steady_clock;
    clock::time_point startTime;
    clock::time_point endTime;
    clock::duration allTime;

    uint64_t kernel_timeMsec;
    uint64_t cpu_timeMsec;


    int N = atoi(argv[1]); // length of input vector
    int M = atoi(argv[2]); // length of filter vector
    int R = N+M-1; // resulting vector size
    R = next_power_two(R); // bumping R to the next power of two


    // initialize host vectors h_I (input), h_F (filter), and h_R (result)
    VectorComplex h_I, h_F, h_R;
    h_I.length = h_F.length = h_R.length = R;
    size_t size_h = R * sizeof(hipfftComplex);


    // allocate memory on host for h_I, h_F, and h_R
    h_I.elements = (hipfftComplex*)malloc(size_h);
    h_F.elements = (hipfftComplex*)malloc(size_h);
    h_R.elements = (hipfftComplex*)malloc(size_h);


    // fill real vectors of h_I and h_F on host
    if (N >= M) {
        for (int i=0; i<N; i++){
            // h_I.elements[i] = sin(i);
            h_I.elements[i].x = i;
            if (i < M) {
                // h_F.elements[i] = rand() % 10;
                h_F.elements[i].x = 1;
            }
        }
        // zero pad remaining values in h_I and h_F
        for (int i = N; i < R; i++) {
            h_I.elements[i].x = 0;
            h_F.elements[i].x = 0;
        }
    } else {
        printf("Input length N (%d) greater than filter length M (%d)...\n", N, N);
        return 0;
    }

    // filling imaginary vectors of h_I, h_F, and h_R with zeros
    for (int i = 0; i < R; i++) {
        h_I.elements[i].y = 0;
        h_F.elements[i].y = 0;
        h_R.elements[i].y = 0;
    }
    // printf("INPUT VECTORS\n");
    // printf("h_I (input)\n");
    // print_complex_vector(h_I.elements, R);
    // printf("h_F (filter)\n");
    // print_complex_vector(h_F.elements, R);
    // printf("\n\n");

    // allocate device memory for d_I, d_F, and d_R
    VectorComplex d_I, d_F, d_R;
    d_I.length = d_F.length = d_R.length = R;
    size_t size_d = R * sizeof(hipfftComplex);

// ####################################### FFT ON DEVICE #######################################
    startTime = clock::now();

    hipMalloc(&d_I.elements, size_d);
    hipMalloc(&d_F.elements, size_d);
    hipMalloc(&d_R.elements, size_d);

    // Copy data from host to device
    hipMemcpy(d_I.elements, h_I.elements, size_d, hipMemcpyHostToDevice);
    hipMemcpy(d_F.elements, h_F.elements, size_d, hipMemcpyHostToDevice);

    // Plan for CUBLAS fft implementation
    hipfftHandle plan;
    hipfftPlan1d(&plan, R, HIPFFT_C2C, 1);

    // Execute FFT on h_I and h_F
    hipfftExecC2C(plan, d_I.elements, d_I.elements, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_F.elements, d_F.elements, HIPFFT_FORWARD);

// ################################# POINT WISE MULT ON DEVICE #################################

    // updated fft values already stored on device in d_I, d_F, and d_R
    PointWiseMult(d_I.elements, d_F.elements, d_R.elements, R);
    hipDeviceSynchronize();
// ####################################### IFFT ON DEVICE ######################################

    // updated mult result values already stored on device in d_R
    hipfftExecC2C(plan, d_R.elements, d_R.elements, HIPFFT_BACKWARD);

// ##################################### COPY BACK TO HOST #####################################

    // copy results back to host
    hipMemcpy(h_R.elements, d_R.elements, size_d, hipMemcpyDeviceToHost);

    endTime = clock::now();
    allTime = endTime - startTime;

    // printf("DEVICE OUTPUT VECTOR\n");
    // printf("h_R (device result)\n");
    // print_complex_vector(h_R.elements, R);

    // device conv timing results
    kernel_timeMsec = std::chrono::duration_cast<std::chrono::microseconds>(allTime).count();
    printf("%8lu ms to convolve two a length %d and length %d complex array on DEVICE.\n", kernel_timeMsec, N, M);
    printf("\n\n");

// ########################### ERROR CHECKING AND CPU COMPARISON ###############################

    VectorComplex cpu_R;
    cpu_R.length = R;
    cpu_R.elements = (hipfftComplex*)malloc(size_h);

    startTime = clock::now();

    fft(h_I.elements, R); // perform fft on h_I
    fft(h_F.elements, R); // perform fft on h_F

    // complex multiplication of fft(h_I) and fft(h_F)
    for (int i = 0; i < R; i++) {
        float a = h_I.elements[i].x; 
        float b = h_I.elements[i].y;
        float c = h_F.elements[i].x; 
        float d = h_F.elements[i].y;

        cpu_R.elements[i].x = ((a * c) - (b * d)) / R; // scale by 1/R
        cpu_R.elements[i].y = ((a * d) + (b * c)) / R; // scale by 1/R
    }

    ifft(cpu_R.elements, R);

    endTime = clock::now();
    allTime = endTime - startTime;

    // printf("HOST OUTPUT VECTOR\n");
    // printf("cpu_R (host result)\n");
    // print_complex_vector(cpu_R.elements, R);
    // printf("\n\n");

    // host conv timing results
    cpu_timeMsec = std::chrono::duration_cast<std::chrono::microseconds>(allTime).count();
    printf("%8lu ms to convolve two a length %d and length %d complex array on HOST.\n", cpu_timeMsec, N, M);
    printf("\n\n");


    // doesn't handle float outputs very well so need to round of find another way to validate

    // // error checking
    // int error_count = 0;
    // for (int i = 0; i < R; i++){
    //     if ((h_R.elements[i].x != cpu_R.elements[i].x) || (h_R.elements[i].y != cpu_R.elements[i].y)){
    //         error_count += 1;   
    //         printf("ERROR in element #%d -> GPU: (%f+j%f) | CPU: (%f+j%f)\n", i, h_R.elements[i].x, h_R.elements[i].y, cpu_R.elements[i].x, cpu_R.elements[i].y);
    //     }
    // }
    // if (error_count != 0) {
    //     printf("Total errors = %d", error_count);
    // }

// #############################################################################################

    // Free host memory
    free(h_I.elements);
    free(h_F.elements);
    free(h_R.elements);
    free(cpu_R.elements);

    // Free device memory
    hipFree(d_I.elements);
    hipFree(d_F.elements);
    hipFree(d_R.elements);

    // Destroy FFT plan
    hipfftDestroy(plan);

    return 0;
}

