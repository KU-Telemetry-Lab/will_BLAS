#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>

#include "kernel_1.h"
#define FILTER_LENGTH 512


__constant__ int F[FILTER_LENGTH];

__global__ void ConvKernel(int* I, int* R, int filter_length, int padded_length) {
    // determine thread id
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // check boundary of the resulting vector
    if (thread_id >= padded_length) return;

    int temp = 0;        
    // step through each element in the filter
    for (int i = 0; i < filter_length; i++) {
        // calculate the corresponding index in the input vector
        int inputIdx = thread_id - i;
        // check if the index is within the bounds of the input vector
        if (inputIdx >= 0 && inputIdx < padded_length) {
            temp += I[inputIdx] * F[i];
        }
    }
    // write back results
    R[thread_id] = temp;
}


void Conv(int* I, int *R, int filter_length, int padded_length) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (padded_length + threadsPerBlock -1) / threadsPerBlock;

    ConvKernel<<<blocksPerGrid, threadsPerBlock>>>(I, R, filter_length, padded_length);
}


void FillConstant(int* I) {
    hipMemcpyToSymbol(HIP_SYMBOL(F), I, FILTER_LENGTH * sizeof(int));
}